#include <stdio.h>
#include <hip/hip_runtime.h>

#define GRID_SIZE 2
#define BLOCK_SIZE 2

__global__ void addvs(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < GRID_SIZE * BLOCK_SIZE)
        c[idx] = a[idx] + b[idx];
}

int main()
{
    int *a, *b, *c, *d_a, *d_b, *d_c;
    int size = GRID_SIZE * BLOCK_SIZE * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    for (int i = 0; i < GRID_SIZE * BLOCK_SIZE; i++)
    {
        a[i] = 2 * i;
        b[i] = 3 * i;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    addvs<<<GRID_SIZE, BLOCK_SIZE>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < GRID_SIZE * BLOCK_SIZE; i++)
        printf("%d ", c[i]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
}
