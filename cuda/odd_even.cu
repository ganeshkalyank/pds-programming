#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void oddeven(int *x, int I, int n)
{
    int id = blockIdx.x;
    if (I == 0 && ((id * 2 + 1) < n))
    {
        if (x[id * 2] > x[id * 2 + 1])
        {
            int X = x[id * 2];
            x[id * 2] = x[id * 2 + 1];
            x[id * 2 + 1] = X;
        }
    }
    if (I == 1 && ((id * 2 + 2) < n))
    {
        if (x[id * 2 + 1] > x[id * 2 + 2])
        {
            int X = x[id * 2 + 1];
            x[id * 2 + 1] = x[id * 2 + 2];
            x[id * 2 + 2] = X;
        }
    }
}

int main()
{
    int a[100], n, c[100], i;
    int *d;

    printf("Enter how many elements of first array:");
    scanf("%d", &n);
    printf("Enter No.\n");
    for (i = 0; i < n; i++)
    {
        scanf("%d", &a[i]);
    }

    hipMalloc((void **)&d, n * sizeof(int));

    hipMemcpy(d, a, n * sizeof(int), hipMemcpyHostToDevice);

    for (i = 0; i < n; i++)
    {

        // int size=n/2;

        oddeven<<<n / 2, 1>>>(d, i % 2, n);
    }
    printf("\n");

    hipMemcpy(c, d, n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Sorted Array is:\t");
    for (i = 0; i < n; i++)
    {
        printf("%d\t", c[i]);
    }

    hipFree(d);
    return 0;
}
