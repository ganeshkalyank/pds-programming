#include <stdio.h>
#include <hip/hip_runtime.h>

#define GRID_SIZE 4
#define BLOCK_SIZE 4

__global__ void product(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < GRID_SIZE * BLOCK_SIZE)
        c[idx] = a[idx] * b[idx];
}

__global__ void sum(int *c, int *partial_sums)
{
    extern __shared__ int sdata[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;
    if (idx < GRID_SIZE * BLOCK_SIZE)
        sdata[tid] = c[idx];
    else
        sdata[tid] = 0;

    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (tid < i)
            sdata[tid] += sdata[tid + i];
        __syncthreads();
    }

    if (tid == 0)
        partial_sums[blockIdx.x] = sdata[0];
}

int main()
{
    int *a, *b, *d_a, *d_b, *d_c, *partial_sums, *d_partial_sums;
    int size = GRID_SIZE * BLOCK_SIZE * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    partial_sums = (int *)malloc(GRID_SIZE * sizeof(int));

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    hipMalloc(&d_partial_sums, GRID_SIZE * sizeof(int));

    for (int i = 0; i < GRID_SIZE * BLOCK_SIZE; i++)
    {
        a[i] = 2 * i;
        b[i] = 3 * i;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    product<<<GRID_SIZE, BLOCK_SIZE>>>(d_a, d_b, d_c);
    sum<<<GRID_SIZE, BLOCK_SIZE>>>(d_c, d_partial_sums);

    hipMemcpy(partial_sums, d_partial_sums, GRID_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    int ans = 0;
    for (int i = 0; i < BLOCK_SIZE; i++)
        ans += partial_sums[i];

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    printf("Dot product: %d\n", ans);

    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Elapsed Time: %f ms\n", elapsedTime);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
}
