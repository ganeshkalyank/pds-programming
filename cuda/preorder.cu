#include <stdio.h>
#include <hip/hip_runtime.h>
__device__ int temp[9][9];
__global__ void traverse(int *parent, int *child, int *sibling, int *edge0, int *edge1, int *succ0, int *succ1, int *position, int *preorder)
{
    int i = threadIdx.x;
    if (parent[edge0[i]] == edge1[i])
    {                                // upward edge
        if (sibling[edge0[i]] != -1) // sibling exists
        {
            succ0[i] = edge1[i];
            succ1[i] = sibling[edge0[i]];
        }
        else if (parent[edge1[i]] != -1) // parent exists
        {
            succ0[i] = edge1[i];
            succ1[i] = parent[edge1[i]];
        }
        else // when no parent and no sibling -> root
        {
            succ0[i] = edge0[i];
            succ1[i] = edge1[i];
            preorder[edge1[i]] = 1; // position for root in preorder
        }
    }
    else // downward edge
    {
        if (child[edge1[i]] != -1) // child exists
        {
            succ0[i] = edge1[i];
            succ1[i] = child[edge1[i]];
        }
        else // No child
        {
            succ0[i] = edge1[i];
            succ1[i] = edge0[i];
        }
    }
    __syncthreads();

    // asign positions for each node
    if (parent[edge0[i]] == edge1[i]) // upward edge
    {
        position[i] = 0;
    }
    else // downward edge
    {
        position[i] = 1;
    }
    __syncthreads();

    int x;
    // list ranking algorithm
    for (int k = 0; k < 4; k++)
    {
        x = temp[succ0[i]][succ1[i]];
        position[i] = position[i] + position[x];
        succ0[i] = succ0[x];
        succ1[i] = succ1[x];
        __syncthreads();
    }

    // preorder position
    if (edge0[i] == parent[edge1[i]])
    {
        preorder[edge1[i]] = 9 + 1 - position[i];
    }
}

// initializing temp
__global__ void initialize(int *edge0, int *edge1)
{
    for (int i = 0; i < 16; i++)
    {
        temp[edge0[i]][edge1[i]] = i;
    }
}

int main()
{
    // input the binary tree
    char vertices[9] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i'};
    int parent[9] = {-1, 0, 0, 1, 1, 2, 3, 3, 4};
    // add only first child
    int child[9] = {1, 3, 5, 6, 8, -1, -1, -1, -1};
    // add only right sibling
    int sibling[9] = {-1, 2, -1, 4, -1, -1, 7, -1, -1};
    // add downward edge
    int edge0[16] = {0, 1, 1, 3, 3, 6, 3, 7, 1, 4, 0, 2, 4, 8, 2, 5};
    // add upward edge
    int edge1[16] = {1, 0, 3, 1, 6, 3, 7, 3, 4, 1, 2, 0, 8, 4, 5, 2};
    int preorder[9];

    // device variables
    int *dparent, *dchild, *dsibling, *dedge0, *dedge1, *dsucc0, *dsucc1, *dposition, *dpreorder;

    // cuda event for time calculation
    hipEvent_t stop, start;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // memory allocation for device variables
    hipMalloc((void **)&dparent, 9 * sizeof(int));
    hipMalloc((void **)&dchild, 9 * sizeof(int));
    hipMalloc((void **)&dsibling, 9 * sizeof(int));
    hipMalloc((void **)&dedge0, 16 * sizeof(int));
    hipMalloc((void **)&dedge1, 16 * sizeof(int));
    hipMalloc((void **)&dsucc0, 16 * sizeof(int));
    hipMalloc((void **)&dsucc1, 16 * sizeof(int));
    hipMalloc((void **)&dposition, 16 * sizeof(int));
    hipMalloc((void **)&dpreorder, 9 * sizeof(int));

    // copy the tree input to device memory
    hipMemcpy(dparent, &parent, 9 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dchild, &child, 9 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dsibling, &sibling, 9 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dedge0, &edge0, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dedge1, &edge1, 16 * sizeof(int), hipMemcpyHostToDevice);

    // record the start time
    hipEventRecord(start, 0);

    // kernel call
    initialize<<<1, 1>>>(dedge0, dedge1);
    traverse<<<1, 16>>>(dparent, dchild, dsibling, dedge0, dedge1, dsucc0, dsucc1, dposition, dpreorder);

    // record the end time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // copy the results from device memory to host memory
    hipMemcpy(&preorder, dpreorder, 9 * sizeof(int), hipMemcpyDeviceToHost);

    // print the results
    printf("Preorder Traversal numbering to the vertices: \n");
    for (int i = 0; i < 9; i++)
    {
        printf("%c -> %d\n", vertices[i], preorder[i]);
    }

    // calculate and print the elapsed time
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("\nTime %f ms \n", time);

    // deallocate the memory space
    hipFree(dparent);
    hipFree(dchild);
    hipFree(dsibling);
    hipFree(dedge0);
    hipFree(dedge1);
    hipFree(dsucc0);
    hipFree(dsucc1);
    hipFree(dposition);
    hipFree(dpreorder);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}