
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void merge_arrays(int *arr, int n)
{
    int i = threadIdx.x;
    int low, high, index;
    if (i < n / 2)
    {
        low = n / 2;
        high = n - 1;
    }
    else
    {
        low = 0;
        high = n / 2 - 1;
    }
    int x = arr[i];
    while (low < high)
    {
        index = (low + high) / 2;
        if (x < arr[index])
        {
            high = index - 1;
        }
        else
        {
            low = index + 1;
        }
    }
    arr[high + i - n / 2 - 1] = x;
}

int main()
{
    int *arr = (int *)malloc(10 * sizeof(int));
    for (int i = 0; i < 5; i++)
    {
        arr[i] = i + 1;
        arr[i + 5] = i + 2;
    }
    for (int i = 0; i < 10; i++)
    {
        printf("%d ", arr[i]);
    }
    printf("\n");
    int *arr_d;
    hipMalloc(&arr_d, 10 * sizeof(int));
    hipMemcpy(arr_d, arr, 10 * sizeof(int), hipMemcpyHostToDevice);
    merge_arrays<<<1, 10>>>(arr_d, 10);
    hipMemcpy(arr, arr_d, 10 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++)
    {
        printf("%d ", arr[i]);
    }
    printf("\n");
}
