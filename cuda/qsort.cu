
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8
struct bound
{
    int low;
    int high;
};
__device__ struct bound stack[10];
__device__ int top = -1;
__device__ void push(int l, int h)
{
    if (top < N + 1)
    {
        struct bound b;
        b.low = l;
        b.high = h;
        stack[++top] = b;
    }
    // printf("\npushing ->%d, %d", stack[top].low, stack[top].high);
}
__device__ struct bound pop(int pid)
{
    struct bound popVal;
    popVal = stack[top];
    top--;
    return popVal;
}
__global__ void initialize()
{
    push(0, N - 1);
}
__device__ void swap(int *a, int *b)
{
    int t = *a;
    *a = *b;
    *b = t;
}
__device__ void printarr(int *arr)
{
    printf("\n");
    for (int i = 0; i < N; i++)
        printf(" %d\t ", arr[i]);
    printf("\n");
}
__device__ int partition(int *a, int start, int end)
{
    int pivot = a[start], p1 = start + 1, i, temp;
    for (i = start + 1; i <= end; i++)
    {
        if (a[i] < pivot)
        {
            if (i != p1)
            {
                temp = a[p1];
                a[p1] = a[i];
                a[i] = temp;
            }
            p1++;
        }
    }

    a[start] = a[p1 - 1];
    a[p1 - 1] = pivot;
    // printarr(a);
    return p1 - 1;
}
__device__ int sorted = 0;
__global__ void quicksort(int *arr)
{

    int pid = threadIdx.x;
    struct bound b;
    while (sorted < N)
    {
        if (stack[top].low == pid)
        {
            b = pop(pid);
        }
        while (b.low < b.high)
        {
            // printf("\n%d = %d, %d ", pid,  b.low, b.high );
            int med = partition(arr, b.low, b.high);
            // printf("\n median = %d " , med);
            if (med + 1 < b.high)
            {
                push(med + 1, b.high);
            }
            b.high = med - 1;
            if (b.low == b.high)
                atomicAdd(&sorted, 2);
            else
                atomicAdd(&sorted, 1);
        }
        if (b.low == b.high)
            atomicAdd(&sorted, 1);
        __syncthreads();
    }
    __syncthreads();
    // printarr(arr);
    // printf("\nfinal-> %d", sorted);
}
int main()
{

    int arr[N] = {4, 8, 7, 5, 1, 2, 3, 6};
    int *darr;
    hipMalloc((int **)&darr, N * sizeof(int));
    initialize<<<1, 1>>>();
    hipMemcpy(darr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    quicksort<<<1, N>>>(darr);
    hipDeviceSynchronize();
    hipMemcpy(arr, darr, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nQuicksort result: ");
    for (int i = 0; i < N; i++)
    {
        printf("%d, ", arr[i]);
    }
}