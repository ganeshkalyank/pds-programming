#include <hip/hip_runtime.h>
#include <stdio.h>

#define ROWS 4
#define COLS 4

__global__ void transpose(int *a, int *b)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int from, to;
    if (x < ROWS && y < COLS)
    {
        from = x + y * ROWS;
        to = y + x * ROWS;
        b[to] = a[from];
    }
}

int main()
{
    int *a, *b, *dA, *dB;
    a = (int *)calloc(ROWS * COLS, sizeof(int));
    b = (int *)calloc(ROWS * COLS, sizeof(int));

    hipMalloc(&dA, ROWS * COLS * sizeof(int));
    hipMalloc(&dB, ROWS * COLS * sizeof(int));

    for (int i = 0; i < ROWS; i++)
    {
        for (int j = 0; j < COLS; j++)
        {
            scanf("%d", a + i * COLS + j);
        }
    }

    hipMemcpy(dA, a, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice);

    transpose<<<dim3(2, 2), dim3(2, 2)>>>(dA, dB);

    hipMemcpy(b, dB, ROWS * COLS * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < ROWS; i++)
    {
        for (int j = 0; j < COLS; j++)
        {
            printf("%d ", b[i * COLS + j]);
        }
        printf("\n\n");
    }

    return 0;
}
