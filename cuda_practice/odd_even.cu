#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void odd_even(int *a, int n, int i)
{
    int j, k;
    int x = blockIdx.x * 2;
    if (i % 2 == 0 && x < n - 1)
    {
        j = a[x];
        k = a[x + 1];
        a[x] = min(j, k);
        a[x + 1] = max(j, k);
    }
    if (i % 2 == 1 && x < n - 2)
    {
        j = a[x + 1];
        k = a[x + 2];
        a[x + 1] = min(j, k);
        a[x + 2] = max(j, k);
    }
}

int main()
{
    int n, *a, *dA;
    scanf("%d", &n);
    a = (int *)calloc(n, sizeof(int));
    hipMalloc(&dA, n * sizeof(int));

    for (int i = 0; i < n; i++)
    {
        scanf("%d", &a[i]);
    }

    hipMemcpy(dA, a, n * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < n; i++)
    {
        odd_even<<<n / 2, 1>>>(dA, n, i);
    }

    hipMemcpy(a, dA, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
    {
        printf("%d ", a[i]);
    }
    printf("\n");

    return 0;
}
