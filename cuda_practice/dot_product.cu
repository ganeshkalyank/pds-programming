#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void dotProduct(int *a, int *b, int *c, int n, int *total_sum)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
    {
        c[idx] = a[idx] * b[idx];
    }
    atomicAdd(total_sum, c[idx]);
}

int main()
{
    int *a, *b, *c, *dA, *dB, *dC;
    int n;

    scanf("%d", &n);
    a = (int *)calloc(n, sizeof(int));
    b = (int *)calloc(n, sizeof(int));
    c = (int *)calloc(n, sizeof(int));
    for (int i = 0; i < n; i++)
    {
        scanf("%d", a + i);
    }
    for (int i = 0; i < n; i++)
    {
        scanf("%d", b + i);
    }

    hipMalloc(&dA, n * sizeof(int));
    hipMalloc(&dB, n * sizeof(int));
    hipMalloc(&dC, n * sizeof(int));

    hipMemcpy(dA, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, b, n * sizeof(int), hipMemcpyHostToDevice);

    int *total_sum, *dTotalSum;
    total_sum = (int *)calloc(1, sizeof(int));
    hipMalloc(&dTotalSum, sizeof(int));
    hipMemcpy(&dTotalSum, &total_sum, sizeof(int), hipMemcpyHostToDevice);

    dotProduct<<<10, 10>>>(dA, dB, dC, n, dTotalSum);

    hipMemcpy(c, dC, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(total_sum, dTotalSum, sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
    {
        printf("%d ", c[i]);
    }
    printf("\n");

    printf("%d\n", *total_sum);

    free(a);
    free(b);
    free(c);
    free(total_sum);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dTotalSum);

    return 0;
}
