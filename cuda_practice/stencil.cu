#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void stencil(int *in, int *out, int n, int radius)
{
    int x = blockIdx.x;
    int sum = 0;
    for (int i = -radius; i <= radius; i++)
    {
        if (x + i > 0 && x + i < n)
        {
            sum += in[x + i];
        }
    }
    out[x] = sum;
}

int main()
{
    int n, *a, *b, *dA, *dB;
    scanf("%d", &n);
    a = (int *)calloc(n, sizeof(int));
    b = (int *)calloc(n, sizeof(int));
    hipMalloc(&dA, n * sizeof(int));
    hipMalloc(&dB, n * sizeof(int));

    for (int i = 0; i < n; i++)
    {
        scanf("%d", &a[i]);
    }

    hipMemcpy(dA, a, n * sizeof(int), hipMemcpyHostToDevice);

    stencil<<<n, 1>>>(dA, dB, n, 3);

    hipMemcpy(b, dB, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
    {
        printf("%d ", b[i]);
    }
    printf("\n");

    return 0;
}
